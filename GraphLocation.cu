/*
 * GraphLocation.c
 * ������ ������ �����͵��� �׷������� �����ֱ�
 */

#define _CRT_SECURE_NO_WARNINGS
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "clustering.h"
#include "graphLocation.h"
#include "terminal.h"

#define STR_MAX 1024

#define TERMINAL_X_MAX 100
#define TERMINAL_Y_MAX 100

void figurePointing(const char* csv_name, const Point* data, const int data_arr_max, const int n_cluster)
{
	int i, j;
	int coloring = 1;

	terminalSize(TERMINAL_X_MAX, TERMINAL_Y_MAX);

	for (i = 0; i < data_arr_max; i++)
	{
		for (j = 0; j < n_cluster; j++)
		{
			if (!strcmp(csv_name, "iris.csv"))
				gotoxy(data[i].x * 10, data[i].y * 10);
			else
				gotoxy(data[i].x, data[i].y);

			if (data[i].centroid_num == j)
			{
				textColoring(coloring);
				printf("��");
			}
			coloring++;
		}
		coloring = 1;
	}
	gotoxy(0, TERMINAL_Y_MAX - 1);
	textColoring(15);
}